#include "ipc_simulation.h"

#include <thrust/transform.h>
#include <thrust/execution_policy.h>

#include <cstdio>
#include <vector>

void advance_ipc()
{
    double* d_in;
    double* d_out;
    hipMalloc(&d_in, 2324*sizeof(double)); 
    hipMalloc(&d_out, sizeof(double));

    std::vector<double> h_in(2324, 11.);
    hipMemcpy(d_in, h_in.data(), 2324 * sizeof(double), hipMemcpyHostToDevice);

    size_t temp_storage_bytes;
    hipcub::DeviceReduce::Reduce(nullptr, temp_storage_bytes, d_in, d_out, 2324, hipcub::Min(), 11.);

    double* temp_storage;
    hipMalloc(&temp_storage, temp_storage_bytes*sizeof(double)); 

    double result;
    hipcub::DeviceReduce::Reduce(
        temp_storage, temp_storage_bytes, d_in, d_out, 2324, hipcub::Min(), 11.);
    hipMemcpy(&result, d_out, sizeof(double), hipMemcpyDeviceToHost);

    printf("!!!!!!! %.20f\n", result);
}

