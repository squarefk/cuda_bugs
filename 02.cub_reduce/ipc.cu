#include "ipc_simulation.h"

#include <thrust/transform.h>
#include <thrust/execution_policy.h>
#include <vector>

int main()
{
    double* d_in;
    double* d_out;
    hipMalloc(&d_in, 2023*sizeof(double)); 
    hipMalloc(&d_out, sizeof(double));

    std::vector<double> h_in(2023, 11.);
    hipMemcpy(d_in, h_in.data(), 2023 * sizeof(double), hipMemcpyHostToDevice);

    size_t temp_storage_bytes;
    hipcub::DeviceReduce::Reduce(nullptr, temp_storage_bytes, d_in, d_out, 2023, hipcub::Min(), 12.);

    double* temp_storage;
    hipMalloc(&temp_storage, temp_storage_bytes*sizeof(double)); 

    double result;
    hipcub::DeviceReduce::Reduce(
        temp_storage, temp_storage_bytes, d_in, d_out, 2023, hipcub::Min(), 12.);
    hipMemcpy(&result, d_out, sizeof(double), hipMemcpyDeviceToHost);

    advance_ipc<TestSpace>();
    return 0;
}
